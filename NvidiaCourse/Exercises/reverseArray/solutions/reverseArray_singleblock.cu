#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */


// includes, system
#include <stdio.h>
#include <assert.h>

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

// Part 1 of 1: implement the kernel
__global__ void reverseArrayBlock(int *d_out, int *d_in)
{
    int in = threadIdx.x;
    int out = blockDim.x - 1 - threadIdx.x;
    d_out[out] = d_in[in];
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    // pointer for host memory and size
    int *h_a;
    int dimA = 256;

    // pointer for device memory
    int *d_b, *d_a;

    // define grid and block size
    int numBlocks = 1;
    int numThreadsPerBlock = dimA;

    // allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int *) malloc(memSize);
    hipMalloc( (void **) &d_a, memSize );
    hipMalloc( (void **) &d_b, memSize );

    // Initialize input array on host
    for (int i = 0; i < dimA; ++i)
    {
        h_a[i] = i;
    }

    // Copy host array to device array
    hipMemcpy( d_a, h_a, memSize, hipMemcpyHostToDevice );

    // launch kernel
    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);
    reverseArrayBlock<<< dimGrid, dimBlock >>>( d_b, d_a );

    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");

    // device to host copy
    hipMemcpy( h_a, d_b, memSize, hipMemcpyDeviceToHost );

    // Check for any CUDA errors
    checkCUDAError("memcpy");

    // verify the data returned to the host is correct
    for (int i = 0; i < dimA; i++)
    {
        assert(h_a[i] == dimA - 1 - i );
    }

    // free device memory
    hipFree(d_a);
    hipFree(d_b);

    // free host memory
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}
