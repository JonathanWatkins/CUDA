#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.numIterations1 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */
 
/* Matrix transpose with Cuda 
 * Host code.

 * This example transposes arbitrary-size matrices.  It compares a naive
 * transpose kernel that suffers from non-coalesced writes, to an optimized
 * transpose with fully coalesced memory access and no bank conflicts.  On 
 * a G80 GPU, the optimized transpose can be more than 10x faster for large
 * matrices.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

#define BLOCK_DIM 16

// This kernel is optimized to ensure all global reads and writes are coalesced,
// and to avoid bank conflicts in shared memory.  This kernel is up to 11x faster
// than the naive kernel below.  Note that the shared memory array is sized to 
// (BLOCK_DIM+1)*BLOCK_DIM.  This pads each row of the 2D block in shared memory 
// so that bank conflicts do not occur when threads address the array column-wise.
__global__ void transpose(float *odata, float *idata, int width, int height)
{
	__shared__ float block[BLOCK_DIM][BLOCK_DIM+1];
	
	// read the matrix tile into shared memory
        // load one element per thread from device memory (idata) and store it
        // in transposed order in block[][]
	unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
	unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
	if((xIndex < width) && (yIndex < height))
	{
		unsigned int index_in = yIndex * width + xIndex;
		block[threadIdx.y][threadIdx.x] = idata[index_in];
	}

        // synchronise to ensure all writes to block[][] have completed
	__syncthreads();

	// write the transposed matrix tile to global memory (odata) in linear order
	xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
	yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
	if((xIndex < height) && (yIndex < width))
	{
		unsigned int index_out = yIndex * height + xIndex;
		odata[index_out] = block[threadIdx.x][threadIdx.y];
	}
}


// This naive transpose kernel suffers from completely non-coalesced writes.
// It can be up to 10x slower than the kernel above for large matrices.
__global__ void transpose_naive(float *odata, float* idata, int width, int height)
{
   unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
   
   if (xIndex < width && yIndex < height)
   {
       unsigned int index_in  = xIndex + width * yIndex;
       unsigned int index_out = yIndex + height * xIndex;
       odata[index_out] = idata[index_in]; 
   }
}

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);
extern "C" void computeGold( float* reference, float* idata, 
                         const unsigned int size_x, const unsigned int size_y );

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);

    cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    // size of the matrix
#ifdef __DEVICE_EMULATION__
    const unsigned int size_x = 32;
    const unsigned int size_y = 128;
#else
    const unsigned int size_x = 256;
    const unsigned int size_y = 4096;
#endif
    // size of memory required to store the matrix
    const unsigned int mem_size = sizeof(float) * size_x * size_y;
    
    unsigned int timer;
    cutCreateTimer(&timer);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
  	if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
  		cutilDeviceInit(argc, argv);
  	else
  		hipSetDevice( cutGetMaxGflopsDeviceId() );

    // allocate host memory
    float* h_idata = (float*) malloc(mem_size);
    // initalize the memory
    srand(15235911);
    for( unsigned int i = 0; i < (size_x * size_y); ++i) 
    {
        h_idata[i] = (float) i;    // rand(); 
    }

    // allocate device memory
    float* d_idata;
    float* d_odata;
    cutilSafeCall( hipMalloc( (void**) &d_idata, mem_size));
    cutilSafeCall( hipMalloc( (void**) &d_odata, mem_size));

    // copy host memory to device
    cutilSafeCall( hipMemcpy( d_idata, h_idata, mem_size,
                                hipMemcpyHostToDevice) );

    // setup execution parameters
    dim3 grid(size_x / BLOCK_DIM, size_y / BLOCK_DIM, 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);

    // warmup so we don't time CUDA startup
    transpose_naive<<< grid, threads >>>(d_odata, d_idata, size_x, size_y);
    transpose<<< grid, threads >>>(d_odata, d_idata, size_x, size_y);

	// synchronize here, so we make sure that we don't count any time from the asynchronize kernel launches.
	hipDeviceSynchronize();

    int numIterations = 1;

    printf("Transposing a %d by %d matrix of floats...\n", size_x, size_y);

    // execute the kernel
    cutStartTimer(timer);
    for (int i = 0; i < numIterations; ++i)
    {
        transpose_naive<<< grid, threads >>>(d_odata, d_idata, size_x, size_y);
    }
    hipDeviceSynchronize();
    cutStopTimer(timer);
    float naiveTime = cutGetTimerValue(timer);

    // execute the kernel
    
    cutResetTimer(timer);
    cutStartTimer(timer);
    for (int i = 0; i < numIterations; ++i)
    {
        transpose<<< grid, threads >>>(d_odata, d_idata, size_x, size_y);
    }
    hipDeviceSynchronize();
    cutStopTimer(timer);
    float optimizedTime = cutGetTimerValue(timer);

    printf("Naive transpose average time:     %0.3f ms\n", naiveTime / numIterations);
    printf("Optimized transpose average time: %0.3f ms\n\n", optimizedTime / numIterations);

    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    // copy result from device to    host
    float* h_odata = (float*) malloc(mem_size);
    cutilSafeCall( hipMemcpy( h_odata, d_odata, mem_size,
                                hipMemcpyDeviceToHost) );

    // compute reference solution
    float* reference = (float*) malloc( mem_size);

    computeGold( reference, h_idata, size_x, size_y);

    // check result
    CUTBoolean res = cutComparef( reference, h_odata, size_x * size_y);
    printf(    "Test %s\n", (1    == res)    ? "PASSED" : "FAILED");

    // cleanup memory
    free(h_idata);
    free(h_odata);
    free( reference);
    cutilSafeCall(hipFree(d_idata));
    cutilSafeCall(hipFree(d_odata));
    cutilCheckError( cutDeleteTimer(timer));

    hipDeviceReset();
}
